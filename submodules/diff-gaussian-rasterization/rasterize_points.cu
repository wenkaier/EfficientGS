/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor,torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor,torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const int top_k,
	bool enable_loss_weight,
	const torch::Tensor& pix_loss,
	const torch::Tensor& sparse_shs_degree,
	const bool use_hwc,
	const bool extract_gs_info,
	const int reg_loss_k,
	const torch::Tensor& in_gs_info,
	const bool extract_gs_rays_num,
	const int max_gs_num,
	float gs_radius,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color;
  if(use_hwc){
	out_color = torch::full({H, W, NUM_CHANNELS}, 0.0, float_opts);
  }else{
	out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  }
  torch::Tensor out_gs_info;
  if(extract_gs_info){
	out_gs_info = torch::full({H, W, 3}, 0, int_opts);
  }else{
	out_gs_info = torch::full({0}, 0, int_opts);
  }

  torch::Tensor out_reg_loss;
  if(reg_loss_k>=0){
	out_reg_loss = torch::full({H, W, NUM_CHANNELS}, 0.0, float_opts);
  }else{
	out_reg_loss = torch::full({0}, 0.0, float_opts);
  }

  torch::Tensor out_rays_num;
  if(extract_gs_rays_num){
	out_rays_num = torch::full({P*2}, 0, float_opts);
  }else{
	out_rays_num = torch::full({0}, 0, float_opts);
  }

  torch::Tensor out_gs_weight;
  if(max_gs_num>0){
	out_gs_weight = torch::full({H, W, max_gs_num}, 0.0, float_opts);
  }else{
	out_gs_weight = torch::full({0}, 0.0, float_opts);
  }


  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Tensor out_max_weight_pids = torch::full({top_k > 0 ? P : 0}, 0, means3D.options().dtype(torch::kInt32));
  torch::Tensor out_loss_weight = torch::full({enable_loss_weight ? P : 0}, 0, means3D.options().dtype(torch::kFloat32));

  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }

	  bool use_rotmat = rotations.size(1) == 9;
	  bool use_sparse_shs = sparse_shs_degree.size(0) != 0;

	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		use_rotmat,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		out_max_weight_pids.contiguous().data<int>(),
		top_k,
		enable_loss_weight,
		pix_loss.contiguous().data<float>(),
		out_loss_weight.contiguous().data<float>(),
		use_sparse_shs,
		sparse_shs_degree.contiguous().data<int>(),
		extract_gs_info,
		out_gs_info.contiguous().data<int>(),
		reg_loss_k,
		in_gs_info.contiguous().data<int>(),
		out_reg_loss.contiguous().data<float>(),
		extract_gs_rays_num,
		out_rays_num.contiguous().data<float>(),
		max_gs_num,
		out_gs_weight.contiguous().data<float>(),
		gs_radius,
		radii.contiguous().data<int>(),
		use_hwc,
		debug);
  }
  return std::make_tuple(rendered, out_color, out_max_weight_pids, out_gs_info, out_reg_loss,out_loss_weight,out_rays_num, out_gs_weight,radii, geomBuffer, binningBuffer, imgBuffer);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor,torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool use_train_speedup,
	const torch::Tensor& ignored_tiles,
	const torch::Tensor& sparse_shs_degree,
	const int reg_loss_k,
    const torch::Tensor& in_gs_info,
	const torch::Tensor& dL_reg_loss,
	const bool use_hwc,
	const bool debug) 
{
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(use_hwc ? 0 : 1) ;
  const int W = dL_dout_color.size(use_hwc ? 1 : 2) ;
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

	bool use_rotmat = rotations.size(1) == 9;
  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());

  torch::Tensor dL_drotations = torch::zeros({P, use_rotmat ? 9: 4}, means3D.options());
  
  if(P != 0)
  {  
	  bool use_sparse_shs = sparse_shs_degree.size(0) != 0;
	  CudaRasterizer::Rasterizer::backward(P, degree, M, R,
	  background.contiguous().data<float>(),
	  W, H, 
	  means3D.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
	  colors.contiguous().data<float>(),
	  scales.data_ptr<float>(),
	  scale_modifier,
	  use_rotmat,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<float>(),
	  dL_dmeans2D.contiguous().data<float>(),
	  dL_dconic.contiguous().data<float>(),  
	  dL_dopacity.contiguous().data<float>(),
	  dL_dcolors.contiguous().data<float>(),
	  dL_dmeans3D.contiguous().data<float>(),
	  dL_dcov3D.contiguous().data<float>(),
	  dL_dsh.contiguous().data<float>(),
	  dL_dscales.contiguous().data<float>(),
	  dL_drotations.contiguous().data<float>(),
	  use_train_speedup,
	  ignored_tiles.contiguous().data<int>(),
	  use_sparse_shs,
	  sparse_shs_degree.contiguous().data<int>(),
	  reg_loss_k,
	  in_gs_info.contiguous().data<int>(),
	  dL_reg_loss.contiguous().data<float>(),
	  use_hwc,
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}